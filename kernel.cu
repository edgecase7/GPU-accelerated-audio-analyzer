#include "hip/hip_runtime.h"
// kernel.cu
#include <iostream>
#include <fstream>
#include <vector>
#include <hipfft/hipfft.h>
#include "kernel.h"

void perform_fft_on_gpu(double* audio_data, int num_samples, const char* output_filename) {
    // 1. Allocate memory on the GPU
    hipfftDoubleComplex* d_audio_data;
    hipMalloc((void**)&d_audio_data, sizeof(hipfftDoubleComplex) * num_samples);

    // 2. Copy data from Host (CPU) to Device (GPU)
    // Note: cufft requires hipfftDoubleComplex input, so we perform the copy here.
    // For simplicity, this example assumes real input. A full implementation would handle this properly.
    hipMemcpy(d_audio_data, audio_data, sizeof(hipfftDoubleComplex) * num_samples, hipMemcpyHostToDevice);

    // 3. Create a cuFFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, num_samples, HIPFFT_Z2Z, 1); // Z2Z for Complex-to-Complex

    // 4. Execute the FFT
    hipfftExecZ2Z(plan, d_audio_data, d_audio_data, HIPFFT_FORWARD);

    // 5. Copy results from Device back to Host
    std::vector<hipfftDoubleComplex> h_fft_result(num_samples);
    hipMemcpy(h_fft_result.data(), d_audio_data, sizeof(hipfftDoubleComplex) * num_samples, hipMemcpyDeviceToHost);

    // 6. Write results to CSV file
    std::ofstream fout(output_filename);
    for (int i = 0; i < num_samples; ++i) {
        double magnitude = sqrt(h_fft_result[i].x * h_fft_result[i].x + h_fft_result[i].y * h_fft_result[i].y);
        fout << magnitude << "\n";
    }
    fout.close();

    // 7. Clean up
    hipfftDestroy(plan);
    hipFree(d_audio_data);
}